#include <stdio.h>
#include <stdlib.h>
#include "kernel.cu"
#include "support.h"

int main() {
    Timer timer;
    hipError_t cuda_ret;
    int sizes[] = {1500, 1750, 2000};  // Example sizes
    int numSizes = sizeof(sizes) / sizeof(sizes[0]);

    for (int idx = 0; idx < numSizes; ++idx) {
        int N = sizes[idx];
        float *A_h, *B_h, *C_h;
        float *A_d, *B_d, *C_d;
        size_t numElements = N * N;
        size_t size = numElements * sizeof(float);


       A_h = (float*) malloc(size);
        B_h = (float*) malloc(size);
        C_h = (float*) malloc(size);

       for (int i = 0; i < numElements; i++) {
            A_h[i] = (rand() % 100) / 100.0f;
            B_h[i] = (rand() % 100) / 100.0f;
        }

        hipMalloc((void**)&A_d, size);
        hipMalloc((void**)&B_d, size);
        hipMalloc((void**)&C_d, size);

       hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
        hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);

        printf("Running matrix multiplication for size %dx%d\n", N, N);
        startTime(&timer);

        dim3 dimBlock(16, 16);  // Block dimensions
        dim3 dimGrid((N + 15) / 16, (N + 15) / 16);  // Grid dimensions

        cuda_ret = hipDeviceSynchronize();
        stopTime(&timer);
        if(cuda_ret != hipSuccess) printf("Kernel execution failed: %s\n", hipGetErrorString(cuda_ret));
        else printf("Elapsed time: %f ms\n", elapsedTime(timer));

        hipMemcpy(C_h, C_d, size, hipMemcpyDeviceToHost);

        verify(A_h, B_h, C_h, N, N, N);

        free(A_h); free(B_h); free(C_h);
        hipFree(A_d); hipFree(B_d); hipFree(C_d);
    }
    return 0;
}
